#include <hip/hip_runtime.h>
#include <iostream>
#include <map>
#include <random>
#include <time.h>

using namespace std;

__global__ void build_binary_tree(int *x, int *child, int *root,
                                  unsigned int n) {
  unsigned int bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int stride = blockDim.x * gridDim.x;
  unsigned int offset = 0;
  bool newBody = true;
  int rootValue = *root;

  // build binary tree
  int childPath;
  int temp;
  offset = 0;
  while ((bodyIndex + offset) < n) {

    if (newBody) {
      newBody = false;

      temp = 0;
      childPath = 0;
      if (x[bodyIndex + offset] > rootValue) {
        childPath = 1;
      }
    }
    int childIndex = child[temp * 2 + childPath];

    // traverse tree until we hit leaf node
    while (childIndex >= 0) {
      temp = childIndex;
      childPath = 0;
      if (x[bodyIndex + offset] > temp) {
        childPath = 1;
      }

      childIndex = child[2 * temp + childPath];
    }

    if (childIndex != -2) {
      int locked = temp * 2 + childPath;
      if (atomicCAS(&child[locked], childIndex, -2) == childIndex) {
        if (childIndex == -1) {
          child[locked] = x[bodyIndex + offset];
        }

        offset += stride;
        newBody = true;
      }
    }

    __syncthreads(); // not strictly needed
  }
}

int main() {
  unsigned int n = 5;

  int *h_x;
  int *d_x;
  int *h_root;
  int *d_root;
  int *h_child;
  int *d_child;

  // allocate memory
  h_x = new int[n];
  //(int*)malloc(n*sizeof(int))
  h_root = new int();
  //(int*)malloc(sizeof(int));
  h_child = new int[2 * (n + 1)];
  // malloc(2*(n+1)*sizeof(int));
  hipMalloc((void **)&d_root, sizeof(int));
  hipMalloc((void **)&d_x, n * sizeof(int));
  hipMalloc((void **)&d_child, 2 * (n + 1) * sizeof(int));
  hipMemset(d_child, -1, 2 * (n + 1) * sizeof(int));

  std::map<int, int> Map;
  // fill h_temp and h_x arrays
  for (unsigned int i = 0; i < n; i++) {
    h_x[i] = i + 1;
    std::cout << h_x[i];
    Map[i] = i + 1;
  }

  std::cout << endl;

  auto val = Map.find(1);

  auto end = Map.end();
  while (val != Map.end()) {
    auto k = &(*val);
    std::cout << "val " << k << std::endl;

    val++;
  }

  // for(unsigned int i=0;i<n;i++){
  // 	unsigned int j = random() % (n-i);
  // 	int temp = h_x[i];
  // 	h_x[i] = h_x[i+j];
  // 	h_x[i+j] = temp;
  // }
  // h_x[0]=3;
  // h_x[1]=6;
  // h_x[2]=4;
  // h_x[3]=1;
  *h_root = h_x[0];

  for (unsigned int i = 0; i < n; i++) {
    std::cout << h_x[i] << " ";
  }
  std::cout << "" << std::endl;

  // copy data to device
  hipMemcpy(d_root, h_root, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_x, h_x, n * sizeof(int), hipMemcpyHostToDevice);

  // kernel call
  dim3 gridSize = 4;
  dim3 blockSize = 4;
  build_binary_tree<<<gridSize, blockSize>>>(d_x, d_child, d_root, n);

  // copy from device back to host
  hipMemcpy(h_child, d_child, 2 * (n + 1) * sizeof(int),
             hipMemcpyDeviceToHost);

  // cout <<  h_child[0] << endl;
  // print tree

  for (size_t i = 0; i < 2 * (n + 1); i++) {
    std::cout << h_child[i] << "\t";
  };
  // for( auto &i: h_child){

  // };

  // free memory
  delete h_x;
  delete h_root;
  delete h_child;
  hipFree(d_x);
  hipFree(d_root);
  hipFree(d_child);
}