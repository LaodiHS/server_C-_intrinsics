#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>

#include <random>
#include <string>
 
#define DIM 24
 
__device__
void setCell(float* d, int x, int y, float v)
{
    if (x < 0 && x > DIM && y < 0 && y > DIM)
        return; //prevent out of bounds write
    d[y * DIM + x] = v;
}
 
__device__
inline float getCell(float* d, int x, int y)
{
    if (x < 0 && x > DIM && y < 0 && y > DIM)
        return 0;
    else
        return d[y * DIM + x];
}
 
__global__
void compute_main(float* d)
{
    int x = (blockDim.x * blockIdx.x) + threadIdx.x;
    int y = (blockDim.y * blockIdx.y) + threadIdx.y;
 
    int nc = 0;
    int cc = getCell(d, x, y);
 
    nc += getCell(d, x + 1, y);
    nc += getCell(d, x - 1, y);
    nc += getCell(d, x, y - 1);
    nc += getCell(d, x, y + 1);
 
 
    if (nc == 2)
        return;
    if (nc == 3 && cc == 1)
    {
        printf("-"); //!!without these printf() calls, these branches do not work
        setCell(d, x, y, 0);
    }
    if (nc == 3 && cc == 0)
    {
        printf("+");
        setCell(d, x, y, 1);
    }
    else
    {
        if (cc == 1)
        {
            printf("-");
            setCell(d, x, y, 0);
        }
    }
}
 
void draw_output(float* data)
{
    for (int x = 0; x < DIM; ++x)
    {
        for (int y = 0; y < DIM; ++y)
        {
            std::cout << data[y * DIM + x] << " ";
        }
        std::cout << "\n";
    }
}
 
bool* bitBuffer = new bool[DIM * DIM];
void saveOutput(const char* file, float* data, size_t size)
{
    std::ofstream outFile;
    outFile.open(file, std::ios::binary | std::ios::out);
 
    if (!outFile.is_open())
        throw new std::exception("ERROR OPENING OUTPUT FILE");
 
    for (int i = 0; i < (DIM * DIM); ++i)
    {
        bitBuffer[i] = data[i] == 1 ? true : false;
    }
    outFile.write((char*)bitBuffer,sizeof(bool) * (DIM*DIM));
    outFile.close();
}
 
int main()
{
    float* pCPU = new float[DIM * DIM];
    float* pGPU = nullptr;
    hipMalloc(&pGPU, sizeof(float) * (DIM * DIM));
   
    std::random_device rd;
    for (int i = 0; i < (DIM * DIM); ++i)
    {
        pCPU[i] = rd() % 2;
    }
 
    hipMemcpy(pGPU, pCPU, sizeof(float) * (DIM * DIM), hipMemcpyHostToDevice);
 
    dim3 blockSize = dim3(24,24,1);
    dim3 gridSize = dim3(32, 32, 1);
 
    for (int i = 0; i < 1024; ++i)
    {
        compute_main <<<1,blockSize>>> (pGPU);
        hipDeviceSynchronize();
        hipMemcpy(pCPU, pGPU, sizeof(float) * (DIM * DIM), hipMemcpyDeviceToHost);
        std::string fileName = "out\\output-" + std::to_string(i) + ".ca";
        draw_output(pCPU);
        Sleep(2000);
    }
 
    return 0;
}